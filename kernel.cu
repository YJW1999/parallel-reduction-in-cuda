#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#define size 100000 // set the length of input array

__device__ int parallel_reduction(int val, int* smem) {
	smem[threadIdx.x] = val;
	__syncthreads();
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		if (threadIdx.x < (threadIdx.x ^ i)) //idea of XOR to acheive divide and conquer
			smem[threadIdx.x] += smem[threadIdx.x ^ i];
		__syncthreads();
	}

	return smem[0];
}

__global__ void ParallelReductionKernel(const int* input, int* output, int _size)
{
	extern __shared__ int smem[]; //using shared memory

	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + tid;

	int sum = (gid < size) ? input[gid] : 0;

	int blockSum = parallel_reduction(sum, smem);

	if (tid == 0)
	{
		output[blockIdx.x] = blockSum;
	}
}


int main() {
	
	int* arr = new int[size];
	int sum = 0;

	for (int i = 0; i < size; ++i) {
		arr[i] = rand();
		sum += arr[i];
	}

	int* input_data, * output_data;
	hipMalloc((void**)&input_data, sizeof(int) * size);
	hipMalloc((void**)&output_data, sizeof(int) * size);
	hipMemcpy(input_data, arr, size*sizeof(int), hipMemcpyHostToDevice);

	const int blockSize = 256;
	const int gridSize = (size + blockSize - 1) / blockSize;

	ParallelReductionKernel << <gridSize, blockSize>> > (input_data, output_data, size);

	int* output = new int[size];
	hipMemcpy(output, output_data, size*sizeof(int), hipMemcpyDeviceToHost);

	int result = 0;
	for (int i = 0; i < size; ++i) {
		result += output[i];
	}
	
	std::cout << "The expected sum is: " << sum << std::endl;
	std::cout << "The output is: " << result << std::endl;

	delete[] arr;
	delete[] output;
	hipFree(input_data);
	hipFree(output_data);

	return 0;
}
